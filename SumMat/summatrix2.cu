#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

#include "../cuPrintf.cu"

#define NANOSECONDS_PER_SECOND 1E9;

void initializeArray(int*, int);
void stampaMatriceArray(int*, int, int);
void equalArray(int*, int*, int);
void sommaMatriciCPU(int*, int*, int*, int, int);
__global__ void sommaMatriciGPU(int*, int*, int*, int, int);

int main(int argc, char **argv) {
	// numero di blocchi e numero di thread per blocco
	dim3 gridDim, blockDim;
    int N; //numero totale di elementi dell'array
    int num; // radice del numero thread del blocco
	// array memorizzati sull'host
	int *A_host, *B_host, *C_host;
	// array memorizzati sul device
	int *A_device, *B_device, *C_device;
	int *copy; // array in cui copieremo i risultati di C_device
    int size; // size in byte di ciascun array
    int flag;
    hipEvent_t startGPU, stopGPU; // tempi di inizio e fine
    struct timespec startCPU, stopCPU;
    float elapsedCPU, elapsedGPU;
    int numBlocks;
    int threadPerSM;
    const int NUM_SM = 16; // 16 for Fermi
    const int MAX_NUM_THREADS = 1536; // 1536 for Fermi, 2048 for Kepler
    const int MAX_NUM_BLOCKS = 8; // 8 for Fermi, 16 for Kepler
	const int MS_IN_S = 1000;

    if (argc < 4) {
        printf("Numero di parametri insufficiente!\n");
        printf("Uso corretto: %s <NumElementi> <sqrNumThreadPerBlocco> <flagStampa>\n", argv[0]);
        printf("Uso dei valori di default\n");
        N = 256;
        num = 8;
        flag = 0;
    }
    else {
        N = atoi(argv[1]);
        num = atoi(argv[2]);
        flag = atoi(argv[3]);
    }

    blockDim.x = blockDim.y = num;
    numBlocks = MAX_NUM_THREADS / (blockDim.x * blockDim.y);
    threadPerSM = (blockDim.x * blockDim.y) * MAX_NUM_BLOCKS;

    if (flag) {
        printf("Addizione di due matrici quadrate.\n");
        printf("Saranno impiegati %d blocchi di thread.\n", numBlocks);
        printf("Saranno usati %d streaming multiprocessor su %d.\n", numBlocks / MAX_NUM_BLOCKS, NUM_SM);
        if (threadPerSM == MAX_NUM_THREADS) {
            printf("Uso ottimale degli SM!\n");
        }
        else {
            printf("Saranno usati solo %d thread su %d per ogni SM!\n", threadPerSM, MAX_NUM_THREADS);
        }
    }

    // determinazione esatta del numero di blocchi
    gridDim.x = N / blockDim.x + ((N % blockDim.x) == 0? 0: 1);
    gridDim.y = N / blockDim.y + ((N % blockDim.y) == 0? 0: 1);

    // stampa delle info sull'esecuzione del kernel
    if (flag) {
        printf("Numero di elementi = %d\n", N);
        printf("Numero di thread per blocco = %d\n", blockDim.x * blockDim.y);
        printf("Numero di blocchi = %d\n", gridDim.x * gridDim.y);
    }

    // allocazione dati sull'host
    size = sizeof(int) * N * N; // dimensione in byte delle matrici
	A_host = (int *) malloc(size);
	B_host = (int *) malloc(size);
	C_host = (int *) malloc(size);
	copy = (int *) malloc(size);

    // allocazione dati sul device
	hipMalloc((void **) &A_device, size);
	hipMalloc((void **) &B_device, size);
	hipMalloc((void **) &C_device, size);

    // inizializzazione dati sull'host
	initializeArray(A_host, N * N);
	initializeArray(B_host, N * N);

    // copia dei dati dall'host al device
	hipMemcpy(A_device, A_host, size, hipMemcpyHostToDevice);
	hipMemcpy(B_device, B_host, size, hipMemcpyHostToDevice);

    // azzeriamo il contenuto della matrice C
	memset(C_host, 0, size);
	hipMemset(C_device, 0, size);

    // avvia cronometrazione GPU
    hipEventCreate(&startGPU);
    hipEventCreate(&stopGPU);
    hipEventRecord(startGPU);

    // invocazione del kernel
    hipEventRecord(startGPU);
    sommaMatriciGPU<<<gridDim, blockDim>>>(A_device, B_device, C_device, N, N);
    hipEventRecord(stopGPU);

    // ferma cronometrazione GPU
    hipEventSynchronize(stopGPU); // assicura che tutti siano arrivati all'evento stop prima di registrare il tempo
    // tempo tra i due eventi in millisecondi
    hipEventElapsedTime(&elapsedGPU, startGPU, stopGPU);
    hipEventDestroy(startGPU);
    hipEventDestroy(stopGPU);

    // copia dei risultati dal device all'host
	hipMemcpy(copy, C_device, size, hipMemcpyDeviceToHost);

    // invocazione somma seriale
    clock_gettime(CLOCK_REALTIME, &startCPU);
    sommaMatriciCPU(A_host, B_host, C_host, N, N);
	clock_gettime(CLOCK_REALTIME, &stopCPU);
	elapsedCPU = (stopCPU.tv_sec - startCPU.tv_sec) + (stopCPU.tv_nsec - startCPU.tv_nsec) / NANOSECONDS_PER_SECOND;

    // stampa degli array e dei risultati
    if (flag ) {
        printf("array A\n");
        stampaMatriceArray(A_host, N, N);
        printf("array B\n");
        stampaMatriceArray(B_host, N, N);
        printf("Risultati host\n");
        stampaMatriceArray(C_host, N, N);
        printf("Risultati device\n");
        stampaMatriceArray(copy, N, N);
    }

    // test di correttezza
    if (flag) {
        equalArray(copy, C_host, N * N);
    }

    printf("tempo CPU: %.3f ms\n", elapsedCPU * MS_IN_S);
    printf("tempo GPU: %.3f ms\n", elapsedGPU); // already in ms

    // de-allocazione host
	free(A_host);
	free(B_host);
	free(C_host);
	free(copy);

	// de-allocazione device
	hipFree(A_device);
	hipFree(B_device);
	hipFree(C_device);

    return EXIT_SUCCESS;
}

void initializeArray(int *array, int n) {
	int i;

	for (i = 0; i < n; i++)
		array[i] = i;
}

void stampaMatriceArray(int* array, int rows, int cols) {
	int i;

	for (i = 0; i < rows * cols; i++) {
		printf("%6.3d\t", array[i]);

        if (i % cols == cols - 1) {
            printf("\n");
        }
    }
}

void equalArray(int* a, int*b, int n) {
	int i = 0;
	while (a[i] == b[i])
		i++;
	if (i < n)
		printf("I risultati dell'host e del device sono diversi\n");
	else
		printf("I risultati dell'host e del device coincidono\n");
}

// seriale
void sommaMatriciCPU(int *first, int *second, int *result, int rows, int cols) {
    int i;

    for (i = 0; i < rows*cols; i++) {
        result[i] = first[i] + second[i];
    }
}

// parallelo
__global__ void sommaMatriciGPU(int *first, int *second, int *result, int rows, int cols) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    int index = j * gridDim.x * blockDim.x + i;

    if (index < rows*cols) {
        result[index] = first[index] + second[index]; 
    }
}
